#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <assert.h>

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

#include "GpuTimer.h"

#define BLOCK_WIDTH 16

hipError_t testWithCuda(uint8_t *inputImage, uint8_t *outputImage, unsigned int imageWidth, unsigned int imageHeight, unsigned int imageChannels);

__global__ void testKernel(uint8_t *inputImage, uint8_t *outputImage, unsigned int imageWidth, unsigned int imageHeight, unsigned int imageChannels)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int pixelIdx = (imageWidth * y + x) * imageChannels;

	if (x < imageWidth && y < imageHeight) {
		unsigned int subpixelIdx;
		for (unsigned int i = 0; i < imageChannels; i++) {
			subpixelIdx = pixelIdx + i;
			outputImage[subpixelIdx] = inputImage[subpixelIdx];
		}
	}
}

int main(int argc, char *argv[])
{
	if (argc != 2) {
		fprintf(stderr, "Invalid arguments.");
		return 1;
	}
	char *imageName = argv[1];
	cv::Mat inputImage = cv::imread(imageName, CV_LOAD_IMAGE_COLOR);
	if (inputImage.empty()) {
		fprintf(stderr, "Cannot read image file %s.", imageName);
		return 1;
	}
	unsigned int imageSize = inputImage.cols * inputImage.rows * inputImage.channels();
	uint8_t *outputImageData = (uint8_t*)malloc(imageSize * sizeof(uint8_t));

    hipError_t cudaStatus = testWithCuda(inputImage.data, outputImageData, inputImage.cols, inputImage.rows, inputImage.channels());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "testWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	cv::Mat outputImage(inputImage.rows, inputImage.cols, CV_8UC3, outputImageData);
	cv::namedWindow("Input Image", cv::WINDOW_NORMAL);
	cv::namedWindow("Output Image", cv::WINDOW_NORMAL);
	cv::imshow("Input Image", inputImage);
	cv::imshow("Output Image", outputImage);

	cv::waitKey(0);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t testWithCuda(uint8_t *inputImage, uint8_t *outputImage, unsigned int imageWidth, unsigned int imageHeight, unsigned int imageChannels)
{
	GpuTimer cudaMallocInputTimer;
	GpuTimer cudaMallocOutputTimer;
	GpuTimer cudaMemcpyInputTimer;
	GpuTimer cudaKernelTimer;
	GpuTimer cudaMemcpyOutputTimer;
	unsigned int imageSize = imageWidth * imageHeight * imageChannels;
	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for two vectors (one input, one output).
	uint8_t *d_InputImage;
	cudaMallocInputTimer.Start();
    cudaStatus = hipMalloc((void**)&d_InputImage, imageSize * sizeof(uint8_t));
	cudaMallocInputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc (input image) failed!");
        goto Error;
    }
	printf("Time to allocate input:\t\t\t\t%f ms\n", cudaMallocInputTimer.Elapsed());

	
	uint8_t *d_OutputImage;
	cudaMallocOutputTimer.Start();
    cudaStatus = hipMalloc((void**)&d_OutputImage, imageSize * sizeof(uint8_t));
	cudaMallocOutputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc (output image) failed!");
        goto Error;
    }
	printf("Time to allocate output:\t\t\t%f ms\n", cudaMallocOutputTimer.Elapsed());

    // Copy input vectors from host memory to GPU buffers.
	cudaMemcpyInputTimer.Start();
    cudaStatus = hipMemcpy(d_InputImage, inputImage, imageSize * sizeof(uint8_t), hipMemcpyHostToDevice);
	cudaMemcpyInputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy (host -> device) failed!");
        goto Error;
    }
	printf("Time to copy input from host to device:\t\t%f ms\n", cudaMemcpyInputTimer.Elapsed());

    // Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 dimGrid((imageWidth - 1) / BLOCK_WIDTH + 1, (imageHeight - 1) / BLOCK_WIDTH + 1, 1);
	cudaKernelTimer.Start();
    testKernel<<<dimGrid, dimBlock>>>(d_InputImage, d_OutputImage, imageWidth, imageHeight, imageChannels);
	cudaKernelTimer.Stop();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "testKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
	printf("Time to execute kernel:\t\t\t\t%f ms\n", cudaKernelTimer.Elapsed());

    // Copy output vector from GPU buffer to host memory.
	cudaMemcpyOutputTimer.Start();
    cudaStatus = hipMemcpy(outputImage, d_OutputImage, imageSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
	cudaMemcpyOutputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy (device -> host) failed!");
        goto Error;
    }

	{
		printf("Time to copy output from device to host:\t%f ms\n", cudaMemcpyOutputTimer.Elapsed());
	}

Error:
	hipFree(d_InputImage);
    hipFree(d_OutputImage);
    
    return cudaStatus;
}
