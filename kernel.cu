#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <assert.h>
#include <string>
#include <iostream>
#include <fstream>
#include <random>

#define _USE_MATH_DEFINES
#include <math.h>

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

#include "GpuTimer.h"

#define BLOCK_WIDTH 16

hipError_t testWithCuda(uint8_t *inputImage, uint8_t *outputImage, unsigned int imageWidth, unsigned int imageHeight, unsigned int imageChannels);

__global__ void testKernel(uint8_t *inputImage, uint8_t *outputImage, unsigned int imageWidth, unsigned int imageHeight, unsigned int imageChannels)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int pixelIdx = (imageWidth * y + x) * imageChannels;

	if (x < imageWidth && y < imageHeight) {
		unsigned int subpixelIdx;
		for (unsigned int i = 0; i < imageChannels; i++) {
			subpixelIdx = pixelIdx + i;
			outputImage[subpixelIdx] = inputImage[subpixelIdx];
		}
	}
}

void computeRandomSprays_CPU(short int ***spraysX, short int ***spraysY, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfSprays) {
	const unsigned int width = 2 * radius + 1;
	const unsigned int area = width * width;								// compute area of neighborhood
	bool *neighborhood = (bool*)malloc(area * sizeof(bool));				// allocate boolean neighborhood array of size area
	short int *sprayX;														// short integer spray point abscissas array
	short int *sprayY;														// short integer spray point ordinates array
	unsigned int pointIdx;													// sample point index
	float randomRadius;														// random radius
	float randomTheta;														//random theta
	short int randomPointX;													// random point abscissa
	short int randomPointY;													// random point ordinate
	unsigned int randomPointNeighborhoodIdx;								// random point neighborhood index
	std::default_random_engine generator;									// random number generator engine
	std::uniform_real_distribution<float> radiusDistribution(0, radius);	// uniform real distribution for radius in the range (0, radius)
	std::uniform_real_distribution<float> thetaDistribution(0, 2 * M_PI);	// uniform real distribution for theta in the range (0, 2*pi)

	*spraysX = (short int**)malloc(numOfSprays * sizeof(short int*));	// sprays abscissas array
	*spraysY = (short int**)malloc(numOfSprays * sizeof(short int*));	// sprays ordinates array
	
	// initialize neighborbood as empty
	for (unsigned int neighborIdx; neighborIdx < area; neighborIdx++) {
		neighborhood[neighborIdx] = false;
	}

	const unsigned int centerPointNeighborhoodIdx = (width + 1) * radius; // calculate center point neighborhood index
	neighborhood[centerPointNeighborhoodIdx] = true;	// block out upcoming random points from coinciding with the center point

	// spray generation loop
	for (unsigned int sprayIdx = 0; sprayIdx < numOfSprays; sprayIdx++) {
		sprayX = (short int*)malloc(numOfSamplePoints * sizeof(short int));	// allocate spray point abscissas array of size numOfSamplePoints
		sprayY = (short int*)malloc(numOfSamplePoints * sizeof(short int));	// allocate spray point abscissas array of size numOfSamplePoints
		pointIdx = 0;											// reset sample point index to 0
		while(pointIdx < numOfSamplePoints) {					// sample point loop
			randomRadius = radiusDistribution(generator);	// get a random distance from the uniform real distribution for distance
			randomTheta = thetaDistribution(generator);		// get a random theta from the uniform real distribution for theta
			randomPointX = randomRadius * cos(randomTheta);	// compute random point abscissa
			randomPointY = randomRadius * sin(randomTheta);	// compute random point ordinate
			randomPointNeighborhoodIdx = width * (randomPointY + radius) + randomPointX + radius;	//compute random point neighborhood index

			if (!neighborhood[randomPointNeighborhoodIdx]) {		// if the random point is not already a sample point
				neighborhood[randomPointNeighborhoodIdx] = true;	// random point is now in the neighborhood
				sprayX[pointIdx] = randomPointX;					// random point is now a sample point (abscissa)
				sprayY[pointIdx] = randomPointY;					// random point is now a sample point (ordinate)
				pointIdx++;											// advance point index
			}
		}
		(*spraysX)[sprayIdx] = sprayX;				// add resultant spray abscissas to sprays abscissas
		(*spraysY)[sprayIdx] = sprayY;				// add resultant spray ordinates to sprays ordinates
		
		// set neighborhood back to empty
		for (pointIdx = 0; pointIdx < numOfSamplePoints; pointIdx++) {
			randomPointNeighborhoodIdx = width * (sprayY[pointIdx] + radius) + sprayX[pointIdx] + radius;
			neighborhood[randomPointNeighborhoodIdx] = false;	// remove each sample point from neighborhood
		}
	}

	free(neighborhood);	// release allocated memory for neighborhood array
}

cv::Mat generateRandomSprayImage(short int *sprayX, short int *sprayY, const unsigned short int radius, const unsigned int numOfSamplePoints) {
	const unsigned int width = radius * 2 + 1;
	const unsigned int area = width * width;
	uint8_t *neighborhood = (uint8_t*)malloc(area * sizeof(uint8_t));
	for (unsigned int pointIdx = 0; pointIdx < area; pointIdx++) {
		neighborhood[pointIdx] = 0;	// black image
	}
	
	unsigned int pointNeighborhoodIdx;
	for (unsigned int pointIdx = 0; pointIdx < numOfSamplePoints; pointIdx++) {
		pointNeighborhoodIdx = width * (sprayY[pointIdx] + radius) + sprayX[pointIdx] + radius;
		neighborhood[pointNeighborhoodIdx] = 255;	// white pixel where a sample point is present
	}
	
	cv::Mat sprayImage(width, width, CV_8UC1, neighborhood);	//create OpenCV grayscale image from data
	return sprayImage;
}

int main(int argc, char *argv[])
{
	srand(time(NULL));
	const unsigned short int radius = 200;
	const unsigned int numOfSamplePoints = 200;
	const unsigned int numOfSprays = 100;
	short int **spraysX;
	short int **spraysY;
	GpuTimer computeRandomSpraysCPUTimer;
	computeRandomSpraysCPUTimer.Start();
	computeRandomSprays_CPU(&spraysX, &spraysY, radius, numOfSamplePoints, numOfSprays);
	computeRandomSpraysCPUTimer.Stop();

	printf("Time to compute random sprays (CPU): %f ms\n", computeRandomSpraysCPUTimer.Elapsed());

	printf("Writing random sprays (%i) to disk ...\n", numOfSprays);
	char sprayImageName[20];
	for (unsigned int sprayIdx = 0; sprayIdx < numOfSprays; sprayIdx++) {
		cv::Mat sprayImage = generateRandomSprayImage(spraysX[sprayIdx], spraysY[sprayIdx], radius, numOfSamplePoints);
		sprintf(sprayImageName, "spray%i.png", sprayIdx);
		cv::imwrite(sprayImageName, sprayImage);
	}

	if (argc != 2) {
		fprintf(stderr, "Invalid arguments.");
		return 1;
	}
	char *imageName = argv[1];
	cv::Mat inputImage = cv::imread(imageName, CV_LOAD_IMAGE_COLOR);
	if (inputImage.empty()) {
		fprintf(stderr, "Cannot read image file %s.", imageName);
		return 1;
	}
	unsigned int imageSize = inputImage.cols * inputImage.rows * inputImage.channels();
	uint8_t *outputImageData = (uint8_t*)malloc(imageSize * sizeof(uint8_t));

    hipError_t cudaStatus = testWithCuda(inputImage.data, outputImageData, inputImage.cols, inputImage.rows, inputImage.channels());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "testWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	cv::Mat outputImage(inputImage.rows, inputImage.cols, CV_8UC3, outputImageData);
	printf("%s\n", "Writing output image to disk ...");
	cv::imwrite("output.png", outputImage);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t testWithCuda(uint8_t *inputImage, uint8_t *outputImage, unsigned int imageWidth, unsigned int imageHeight, unsigned int imageChannels)
{
	GpuTimer cudaMallocInputTimer;
	GpuTimer cudaMallocOutputTimer;
	GpuTimer cudaMemcpyInputTimer;
	GpuTimer cudaKernelTimer;
	GpuTimer cudaMemcpyOutputTimer;
	unsigned int imageSize = imageWidth * imageHeight * imageChannels;
	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for two vectors (one input, one output).
	uint8_t *d_InputImage;
	cudaMallocInputTimer.Start();
    cudaStatus = hipMalloc((void**)&d_InputImage, imageSize * sizeof(uint8_t));
	cudaMallocInputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc (input image) failed!");
        goto Error;
    }
	printf("Time to allocate input:\t\t\t\t%f ms\n", cudaMallocInputTimer.Elapsed());

	
	uint8_t *d_OutputImage;
	cudaMallocOutputTimer.Start();
    cudaStatus = hipMalloc((void**)&d_OutputImage, imageSize * sizeof(uint8_t));
	cudaMallocOutputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc (output image) failed!");
        goto Error;
    }
	printf("Time to allocate output:\t\t\t%f ms\n", cudaMallocOutputTimer.Elapsed());

    // Copy input vectors from host memory to GPU buffers.
	cudaMemcpyInputTimer.Start();
    cudaStatus = hipMemcpy(d_InputImage, inputImage, imageSize * sizeof(uint8_t), hipMemcpyHostToDevice);
	cudaMemcpyInputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy (host -> device) failed!");
        goto Error;
    }
	printf("Time to copy input from host to device:\t\t%f ms\n", cudaMemcpyInputTimer.Elapsed());

    // Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 dimGrid((imageWidth - 1) / BLOCK_WIDTH + 1, (imageHeight - 1) / BLOCK_WIDTH + 1, 1);
	cudaKernelTimer.Start();
    testKernel<<<dimGrid, dimBlock>>>(d_InputImage, d_OutputImage, imageWidth, imageHeight, imageChannels);
	cudaKernelTimer.Stop();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "testKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
	printf("Time to execute kernel:\t\t\t\t%f ms\n", cudaKernelTimer.Elapsed());

    // Copy output vector from GPU buffer to host memory.
	cudaMemcpyOutputTimer.Start();
    cudaStatus = hipMemcpy(outputImage, d_OutputImage, imageSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
	cudaMemcpyOutputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy (device -> host) failed!");
        goto Error;
    }

	{
		printf("Time to copy output from device to host:\t%f ms\n", cudaMemcpyOutputTimer.Elapsed());
	}

Error:
	hipFree(d_InputImage);
    hipFree(d_OutputImage);
    
    return cudaStatus;
}
