#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <assert.h>
#include <ctime>
#include <string>
#include <iostream>
#include <fstream>
#include <random>

#define _USE_MATH_DEFINES
#include <math.h>

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
//#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "GpuTimer.h"

#define BLOCK_WIDTH 16

hipError_t testWithCuda(uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, const uint8_t imageChannels, short int **spraysX, short int **spraysY, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfSprays, const unsigned int numOfIterations, const unsigned long long seed);

/*__global__ void testKernel(uint8_t *inputImage, uint8_t *outputImage, unsigned int imageWidth, unsigned int imageHeight, unsigned int imageChannels)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int pixelIdx = (imageWidth * y + x) * imageChannels;

	if (x < imageWidth && y < imageHeight) {
		unsigned int subpixelIdx;
		for (unsigned int i = 0; i < imageChannels; i++) {
			subpixelIdx = pixelIdx + i;
			outputImage[subpixelIdx] = inputImage[subpixelIdx];
		}
	}
}*/

void computeRandomSpraysCPU(short int ***spraysX, short int ***spraysY, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfSprays) {
	const unsigned int width = 2 * radius + 1;
	const unsigned int area = width * width;								// compute area of neighborhood
	bool *neighborhood = (bool*)malloc(area * sizeof(bool));				// allocate boolean neighborhood array of size area
	short int *sprayX;														// short integer spray point abscissas array
	short int *sprayY;														// short integer spray point ordinates array
	unsigned int pointIdx;													// sample point index
	float randomRadius;														// random radius
	float randomTheta;														//random theta
	short int randomPointX;													// random point abscissa
	short int randomPointY;													// random point ordinate
	unsigned int randomPointNeighborhoodIdx;								// random point neighborhood index
	std::default_random_engine generator;									// random number generator engine
	std::uniform_real_distribution<float> radiusDistribution(0, radius);	// uniform real distribution for radius in the range (0, radius)
	std::uniform_real_distribution<float> thetaDistribution(0, 2 * M_PI);	// uniform real distribution for theta in the range (0, 2*pi)

	*spraysX = (short int**)malloc(numOfSprays * sizeof(short int*));	// sprays abscissas array
	*spraysY = (short int**)malloc(numOfSprays * sizeof(short int*));	// sprays ordinates array
	
	// initialize neighborbood as empty
	for (unsigned int neighborIdx = 0; neighborIdx < area; neighborIdx++) {
		neighborhood[neighborIdx] = false;
	}

	const unsigned int centerPointNeighborhoodIdx = (width + 1) * radius; // calculate center point neighborhood index
	neighborhood[centerPointNeighborhoodIdx] = true;	// block out upcoming random points from coinciding with the center point

	// spray generation loop
	for (unsigned int sprayIdx = 0; sprayIdx < numOfSprays; sprayIdx++) {
		sprayX = (short int*)malloc(numOfSamplePoints * sizeof(short int));	// allocate spray point abscissas array of size numOfSamplePoints
		sprayY = (short int*)malloc(numOfSamplePoints * sizeof(short int));	// allocate spray point abscissas array of size numOfSamplePoints
		pointIdx = 0;											// reset sample point index to 0
		while(pointIdx < numOfSamplePoints) {					// sample point loop
			randomRadius = radiusDistribution(generator);	// get a random distance from the uniform real distribution for distance
			randomTheta = thetaDistribution(generator);		// get a random theta from the uniform real distribution for theta
			randomPointX = randomRadius * cos(randomTheta);	// compute random point abscissa
			randomPointY = randomRadius * sin(randomTheta);	// compute random point ordinate
			randomPointNeighborhoodIdx = width * (randomPointY + radius) + randomPointX + radius;	//compute random point neighborhood index

			if (!neighborhood[randomPointNeighborhoodIdx]) {		// if the random point is not already a sample point
				neighborhood[randomPointNeighborhoodIdx] = true;	// random point is now in the neighborhood
				sprayX[pointIdx] = randomPointX;					// random point is now a sample point (abscissa)
				sprayY[pointIdx] = randomPointY;					// random point is now a sample point (ordinate)
				pointIdx++;											// advance point index
			}
		}
		(*spraysX)[sprayIdx] = sprayX;				// add resultant spray abscissas to sprays abscissas
		(*spraysY)[sprayIdx] = sprayY;				// add resultant spray ordinates to sprays ordinates
		
		// set neighborhood back to empty
		for (pointIdx = 0; pointIdx < numOfSamplePoints; pointIdx++) {
			randomPointNeighborhoodIdx = width * (sprayY[pointIdx] + radius) + sprayX[pointIdx] + radius;
			neighborhood[randomPointNeighborhoodIdx] = false;	// remove each sample point from neighborhood
		}
	}

	free(neighborhood);	// release allocated memory for neighborhood array
}

cv::Mat generateRandomSprayImage(short int *sprayX, short int *sprayY, const unsigned short int radius, const unsigned int numOfSamplePoints) {
	const unsigned int width = radius * 2 + 1;
	const unsigned int area = width * width;
	uint8_t *neighborhood = (uint8_t*)malloc(area * sizeof(uint8_t));
	for (unsigned int pointIdx = 0; pointIdx < area; pointIdx++) {
		neighborhood[pointIdx] = 0;	// black image
	}
	
	unsigned int pointNeighborhoodIdx;
	for (unsigned int pointIdx = 0; pointIdx < numOfSamplePoints; pointIdx++) {
		pointNeighborhoodIdx = width * (sprayY[pointIdx] + radius) + sprayX[pointIdx] + radius;
		neighborhood[pointNeighborhoodIdx] = 255;	// white pixel where a sample point is present
	}
	
	cv::Mat sprayImage(width, width, CV_8UC1, neighborhood);	//create OpenCV grayscale image from data
	return sprayImage;
}

// This version of the function uses pre-computed random sprays. It chooses, in each iteration, for each pixel, a random spray at random out of the available sprays.
// This introduces an issue whereby pixels closer to the edge of the image in particular face reduced sampling due to many sample points lying outside the image and
// thus not being factored into calculating the envelope. The issue manifests itself particularly when more iterations are used.
void STRESSGrayscaleToGrayscaleCPU1(uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, short int **spraysX, short int **spraysY, const unsigned int numOfSamplePoints, const unsigned int numOfSprays, const unsigned int numOfIterations) {
	unsigned int targetPixelIdx; // target pixel (p) absolute index
	int samplePointX; // spray sample point abscissa
	int samplePointY; // spray sample point ordinate
	unsigned int samplePointPixelIdx; // spray sample point pixel index
	uint8_t Emin;
	uint8_t Emax;

	unsigned int randomSprayIdx;  // random spray index
	short int *randomSprayX;    // abscissas for spray chosen at random
	short int *randomSprayY;    // ordinates for spray chosen at random

	// allocate temporary output image array for storing sum of all iteration results
	unsigned int imageSize = imageWidth * imageHeight;
	float *tempOutputImage = (float*)malloc(imageSize * sizeof(float));

	// initial temporary output image as empty
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		tempOutputImage[pixelIdx] = 0.0f;
	}

	// iteration loop
	for (unsigned int iterationIdx = 0; iterationIdx < numOfIterations; iterationIdx++) {
		targetPixelIdx = 0; // reset target pixel absolute index to 0
		for (unsigned short int targetPixelY = 0; targetPixelY < imageHeight; targetPixelY++) {
			for (unsigned short int targetPixelX = 0; targetPixelX < imageWidth; targetPixelX++) {
				//set Emin and Emax equal to target pixel value
				Emin = Emax = inputImage[targetPixelIdx];

				// choose spray at random
				randomSprayIdx = rand() % numOfSprays;
				randomSprayX = spraysX[randomSprayIdx];
				randomSprayY = spraysY[randomSprayIdx];

				// calculate envelope
				for (unsigned int sampleIdx = 0; sampleIdx < numOfSamplePoints; sampleIdx++) {
					samplePointX = targetPixelX + randomSprayX[sampleIdx];  // get sample point abscissa in input image
					samplePointY = targetPixelY + randomSprayY[sampleIdx];  // get sample point ordinate in input image
																			//printf("%i %i\n", samplePointX, samplePointY);
					if (samplePointX >= 0 && samplePointX < imageWidth && samplePointY >= 0 && samplePointY < imageHeight) {  // only proceed if sample point is within the input image
						samplePointPixelIdx = imageWidth * samplePointY + samplePointX; // get sample point index in input image
						if (inputImage[samplePointPixelIdx] < Emin) // if sample point color channel is less than Emin at that channel
							Emin = inputImage[samplePointPixelIdx]; // it is the new Emin at that channel
						else if (inputImage[samplePointPixelIdx] > Emax)
							Emax = inputImage[samplePointPixelIdx];
					}
				}

				// calculate (p - Emin) / (Emax - Emin)
				tempOutputImage[targetPixelIdx] += (inputImage[targetPixelIdx] - Emin) * 255.0 / (Emax - Emin);

				targetPixelIdx++;
			}
		}
	}

	// divide each accumulated pixel value by the number of iterations to obtain the average pixel value across iterations.
	// place the average value in the output image array.
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		outputImage[pixelIdx] = tempOutputImage[pixelIdx] / numOfIterations;
	}
}

// This version of the function does not use pre-computed random sprays. Instead, it generates, in each iteration, for each pixel in the image, a random spray for that pixel.
// This solves the issue of reduced sampling seen in the first version of the function. However, this approach is much slower than using pre-computed sprays
void STRESSGrayscaleToGrayscaleCPU2(uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfIterations) {
	unsigned int randomSamplePixelIdx;									// random sample pixel index
	unsigned int randomSampleImagePixelIdx;								// random sample pixel absolute index in image
	float randomRadius;													// random radius
	float randomTheta;													// random theta
	int randomSamplePixelX;												// random sample pixel abscissa
	int randomSamplePixelY;												// random sample pixel ordinate
	std::default_random_engine generator;								// random number generator engine
	std::uniform_real_distribution<float> radiusDistribution(0, radius);	// uniform real distribution for radius in the range (0, radius)
	std::uniform_real_distribution<float> thetaDistribution(0, 2 * M_PI);	// uniform real distribution for theta in the range (0, 2*pi)

	unsigned int targetPixelIdx; // target pixel (p) absolute index
	uint8_t Emin;
	uint8_t Emax;

	// allocate temporary output image array for storing sum of all iteration results
	unsigned int imageSize = imageWidth * imageHeight;
	float *tempOutputImage = (float*)malloc(imageSize * sizeof(float));

	// initial temporary output image as empty
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		tempOutputImage[pixelIdx] = 0.0f;
	}

	// iteration loop
	for (unsigned int iterationIdx = 0; iterationIdx < numOfIterations; iterationIdx++) {
		targetPixelIdx = 0;	// reset target pixel absolute index to 0
		for (unsigned short int targetPixelY = 0; targetPixelY < imageHeight; targetPixelY++) {
			for (unsigned short int targetPixelX = 0; targetPixelX < imageWidth; targetPixelX++) {
				//set Emin and Emax equal to target pixel value
				Emin = Emax = inputImage[targetPixelIdx];

				// generate random sample points and calculate envelope
				randomSamplePixelIdx = 0;
				while (randomSamplePixelIdx < numOfSamplePoints) {		// random sample pixel point loop
					randomRadius = radiusDistribution(generator);	// get a random distance from the uniform real distribution for distance
					randomTheta = thetaDistribution(generator);		// get a random theta from the uniform real distribution for theta
					randomSamplePixelX = targetPixelX + randomRadius * cos(randomTheta);	// compute random pixel abscissa
					if (randomSamplePixelX >= 0 && randomSamplePixelX < imageWidth) {		// if random pixel abscissa is within image
						randomSamplePixelY = targetPixelY + randomRadius * sin(randomTheta);		// compute random pixel ordinate
						if (randomSamplePixelY >= 0 && randomSamplePixelY < imageHeight) {	// if random pixel ordinate is within image
							randomSampleImagePixelIdx = imageWidth * randomSamplePixelY + randomSamplePixelX; // get random sample pixel index in image
							if (inputImage[randomSampleImagePixelIdx] < Emin)		// if sample pixel value is less than Emin
								Emin = inputImage[randomSampleImagePixelIdx];		// it is the new Emin
							else if (inputImage[randomSampleImagePixelIdx] > Emax)	// if sample pixel value is greater than Emax 
								Emax = inputImage[randomSampleImagePixelIdx];	// it is the new Emax
							randomSamplePixelIdx++;	// advance random sample pixel index
						}
					}
				}

				// calculate (p - Emin) / (Emax - Emin)
				tempOutputImage[targetPixelIdx] += (inputImage[targetPixelIdx] - Emin) * 255.0 / (Emax - Emin);

				targetPixelIdx++; // advance target pixel index
			}
		}
	}

	// divide each accumulated pixel value by the number of iterations to obtain the average pixel value across iterations.
	// place the average value in the output image array.
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		outputImage[pixelIdx] = tempOutputImage[pixelIdx] / numOfIterations;
	}
}

// This version of the function is a hybrid between the first two approaches. It uses pre-computed sprays similarly to the first approach.
// However, for any pixel, if any sample point in its chosen pre-computed spray is found to be lying outside the image, it is replaced with
// a randomly chosen sample points lying within the image. This should solve the issue of the first approach while not being as slow as the second approach,
// particularly for pixels not close to the edges of the image, since the likelihood of a sample point not lying within the image for those diminishes greatly.
void STRESSGrayscaleToGrayscaleCPU3(uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, short int **spraysX, short int **spraysY, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfSprays, const unsigned int numOfIterations) {
	float randomRadius;													// random radius
	float randomTheta;													// random theta
	int samplePixelX;													// (pre-computed / random) sample pixel abscissa
	int samplePixelY;													// (pre-computed / random) sample pixel ordinate
	unsigned int sampleIdx;												// sample index
	unsigned int samplePixelIdx;										// sample pixel index
	unsigned int numOfValidSamplePoints;								// number of valid sample points in envelope
	std::default_random_engine generator;								// random number generator engine
	std::uniform_real_distribution<float> radiusDistribution(0, radius);	// uniform real distribution for radius in the range (0, radius)
	std::uniform_real_distribution<float> thetaDistribution(0, 2 * M_PI);	// uniform real distribution for theta in the range (0, 2*pi)
	
	unsigned int targetPixelIdx; // target pixel (p) absolute index
	uint8_t Emin;
	uint8_t Emax;

	unsigned int randomSprayIdx;  // random spray index
	short int *randomSprayX;    // abscissas for spray chosen at random
	short int *randomSprayY;    // ordinates for spray chosen at random

								// allocate temporary output image array for storing sum of all iteration results
	unsigned int imageSize = imageWidth * imageHeight;
	float *tempOutputImage = (float*)malloc(imageSize * sizeof(float));

	// initial temporary output image as empty
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		tempOutputImage[pixelIdx] = 0.0f;
	}

	// iteration loop
	for (unsigned int iterationIdx = 0; iterationIdx < numOfIterations; iterationIdx++) {
		targetPixelIdx = 0; // reset target pixel absolute index to 0
		for (unsigned short int targetPixelY = 0; targetPixelY < imageHeight; targetPixelY++) {
			for (unsigned short int targetPixelX = 0; targetPixelX < imageWidth; targetPixelX++) {
				//set Emin and Emax equal to target pixel value
				Emin = Emax = inputImage[targetPixelIdx];

				// choose spray at random
				randomSprayIdx = rand() % numOfSprays;
				randomSprayX = spraysX[randomSprayIdx];
				randomSprayY = spraysY[randomSprayIdx];

				// calculate envelope
				sampleIdx = 0;	// reset sample index to 0
				numOfValidSamplePoints = 0;	// reset number of valid sample points to 0
				for (sampleIdx = 0; sampleIdx < numOfSamplePoints; sampleIdx++) {
					samplePixelX = targetPixelX + randomSprayX[sampleIdx];  // get sample pixel abscissa in input image
					samplePixelY = targetPixelY + randomSprayY[sampleIdx];  // get sample pixel ordinate in input image
					if (samplePixelX >= 0 && samplePixelX < imageWidth && samplePixelY >= 0 && samplePixelY < imageHeight) {  // only proceed if sample pixel is within the input image
						samplePixelIdx = imageWidth * samplePixelY + samplePixelX; // get sample pixel index in input image
						if (inputImage[samplePixelIdx] < Emin) // if sample pixel value is less than Emin
							Emin = inputImage[samplePixelIdx]; // it is the new Emin
						else if (inputImage[samplePixelIdx] > Emax)	// if sample pixel value is greater than Emax
							Emax = inputImage[samplePixelIdx];			// it is the new Emax
						numOfValidSamplePoints++;	// increment number of valid sample points
					}
				}

				// generate sample points to compensate for invalid sample points
				sampleIdx = numOfValidSamplePoints;
				while (sampleIdx < numOfSamplePoints) {
					randomRadius = radiusDistribution(generator);	// get a random distance from the uniform real distribution for distance
					randomTheta = thetaDistribution(generator);		// get a random theta from the uniform real distribution for theta
					samplePixelX = targetPixelX + randomRadius * cos(randomTheta);	// compute random pixel abscissa
					if (samplePixelX >= 0 && samplePixelX < imageWidth) {		// if random pixel abscissa is within image
						samplePixelY = targetPixelY + randomRadius * sin(randomTheta);		// compute random pixel ordinate
						if (samplePixelY >= 0 && samplePixelY < imageHeight) {	// if random pixel ordinate is within image
							samplePixelIdx = imageWidth * samplePixelY + samplePixelX; // get random sample pixel index in image
							if (inputImage[samplePixelIdx] < Emin)		// if sample pixel value is less than Emin
								Emin = inputImage[samplePixelIdx];		// it is the new Emin
							else if (inputImage[samplePixelIdx] > Emax)	// if sample pixel value is greater than Emax 
								Emax = inputImage[samplePixelIdx];	// it is the new Emax
							sampleIdx++;	// advance random sample pixel index
						}
					}
				}

				// calculate (p - Emin) / (Emax - Emin)
				tempOutputImage[targetPixelIdx] += (inputImage[targetPixelIdx] - Emin) * 255.0 / (Emax - Emin);

				targetPixelIdx++;
			}
		}
	}

	// divide each accumulated pixel value by the number of iterations to obtain the average pixel value across iterations.
	// place the average value in the output image array.
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		outputImage[pixelIdx] = tempOutputImage[pixelIdx] / numOfIterations;
	}
}

void STRESSColorToGrayscaleCPU3(uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, const uint8_t imageChannels, short int **spraysX, short int **spraysY, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfSprays, const unsigned int numOfIterations) {
	float randomRadius;													// random radius
	float randomTheta;													// random theta
	int samplePixelX;													// (pre-computed / random) sample pixel abscissa
	int samplePixelY;													// (pre-computed / random) sample pixel ordinate
	unsigned int sampleIdx;												// sample index
	uint8_t channelIdx;													// channel index
	unsigned int samplePixelIdx;										// sample pixel index
	unsigned int samplePixelChannelIdx;									// sample pixel channel index
	unsigned int numOfValidSamplePoints;								// number of valid sample points in envelope
	std::default_random_engine generator;								// random number generator engine
	std::uniform_real_distribution<float> radiusDistribution(0, radius);	// uniform real distribution for radius in the range (0, radius)
	std::uniform_real_distribution<float> thetaDistribution(0, 2 * M_PI);	// uniform real distribution for theta in the range (0, 2*pi)

	unsigned int targetInputPixelIdx; // target input pixel (p) index
	unsigned int targetOutputPixelIdx; // target output pixel index
	uint8_t *Emin = (uint8_t*)malloc(imageChannels * sizeof(uint8_t));	// Emin array of size imageChannels
	uint8_t *Emax = (uint8_t*)malloc(imageChannels * sizeof(uint8_t));	// Emax array of size imageChannels
	
	// for calculating (p - Emin).(Emax - Emin) / |Emax - Emin|^2
	uint8_t Edelta;
	unsigned int dotProd, ElenSq;

	unsigned int randomSprayIdx;  // random spray index
	short int *randomSprayX;    // abscissas for spray chosen at random
	short int *randomSprayY;    // ordinates for spray chosen at random

	// allocate temporary output image array for storing sum of all iteration results
	unsigned int outputImageSize = imageWidth * imageHeight;
	float *tempOutputImage = (float*)malloc(outputImageSize * sizeof(float));

	// initial temporary output image as empty
	for (unsigned int pixelIdx = 0; pixelIdx < outputImageSize; pixelIdx++) {
		tempOutputImage[pixelIdx] = 0.0f;
	}

	// iteration loop
	for (unsigned int iterationIdx = 0; iterationIdx < numOfIterations; iterationIdx++) {
		targetInputPixelIdx = 0; // reset target input pixel index to 0
		targetOutputPixelIdx = 0; // reset target output pixel index to 0
		for (unsigned short int targetPixelY = 0; targetPixelY < imageHeight; targetPixelY++) {
			for (unsigned short int targetPixelX = 0; targetPixelX < imageWidth; targetPixelX++) {
				//set Emin and Emax equal to target pixel across all color channels
				for (channelIdx = 0; channelIdx < imageChannels; channelIdx++)
					Emin[channelIdx] = Emax[channelIdx] = inputImage[targetInputPixelIdx + channelIdx];

				// choose spray at random
				randomSprayIdx = rand() % numOfSprays;
				randomSprayX = spraysX[randomSprayIdx];
				randomSprayY = spraysY[randomSprayIdx];

				// calculate envelope
				sampleIdx = 0;	// reset sample index to 0
				numOfValidSamplePoints = 0;	// reset number of valid sample points to 0
				for (sampleIdx = 0; sampleIdx < numOfSamplePoints; sampleIdx++) {
					samplePixelX = targetPixelX + randomSprayX[sampleIdx];  // get sample pixel abscissa in input image
					samplePixelY = targetPixelY + randomSprayY[sampleIdx];  // get sample pixel ordinate in input image
					if (samplePixelX >= 0 && samplePixelX < imageWidth && samplePixelY >= 0 && samplePixelY < imageHeight) {  // only proceed if sample pixel is within the input image
						samplePixelIdx = (imageWidth * samplePixelY + samplePixelX) * imageChannels; // get sample pixel index in input image
						for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
							samplePixelChannelIdx = samplePixelIdx + channelIdx;
							if (inputImage[samplePixelChannelIdx] < Emin[channelIdx]) // if sample pixel value is less than Emin
								Emin[channelIdx] = inputImage[samplePixelChannelIdx];		// it is the new Emin
							else if (inputImage[samplePixelChannelIdx] > Emax[channelIdx])	// if sample pixel value is greater than Emax
								Emax[channelIdx] = inputImage[samplePixelChannelIdx];		// it is the new Emax
						}
						numOfValidSamplePoints++;	// increment number of valid sample points
					}
				}

				// generate sample points to compensate for invalid sample points
				sampleIdx = numOfValidSamplePoints;
				while (sampleIdx < numOfSamplePoints) {
					randomRadius = radiusDistribution(generator);	// get a random distance from the uniform real distribution for distance
					randomTheta = thetaDistribution(generator);		// get a random theta from the uniform real distribution for theta
					samplePixelX = targetPixelX + randomRadius * cos(randomTheta);	// compute random pixel abscissa
					if (samplePixelX >= 0 && samplePixelX < imageWidth) {		// if random pixel abscissa is within image
						samplePixelY = targetPixelY + randomRadius * sin(randomTheta);		// compute random pixel ordinate
						if (samplePixelY >= 0 && samplePixelY < imageHeight) {	// if random pixel ordinate is within image
							samplePixelIdx = (imageWidth * samplePixelY + samplePixelX) * imageChannels; // get random sample pixel index in image
							for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
								samplePixelChannelIdx = samplePixelIdx + channelIdx;
								if (inputImage[samplePixelChannelIdx] < Emin[channelIdx])		// if sample pixel value is less than Emin
									Emin[channelIdx] = inputImage[samplePixelChannelIdx];		// it is the new Emin
								else if (inputImage[samplePixelChannelIdx] > Emax[channelIdx])	// if sample pixel value is greater than Emax 
									Emax[channelIdx] = inputImage[samplePixelChannelIdx];		// it is the new Emax
							}
							sampleIdx++;	// advance random sample pixel index
						}
					}
				}

				// calculate (p - Emin).(Emax - Emin), |Emax - Emin|^2
				dotProd = 0;
				ElenSq = 0;
				for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
					Edelta = Emax[channelIdx] - Emin[channelIdx];
					dotProd += Edelta * (inputImage[targetInputPixelIdx + channelIdx] - Emin[channelIdx]);
					ElenSq += Edelta * Edelta;
				}

				// calculate g = (p - Emin).(Emax - Emin) / |Emax - Emin|^2
				tempOutputImage[targetOutputPixelIdx] += dotProd * 255.0 / ElenSq;

				targetInputPixelIdx += imageChannels;
				targetOutputPixelIdx++;
			}
		}
	}

	// divide each accumulated pixel value by the number of iterations to obtain the average pixel value across iterations.
	// place the average value in the output image array.
	for (unsigned int pixelIdx1 = 0; pixelIdx1 < outputImageSize; pixelIdx1++) {
		outputImage[pixelIdx1] = tempOutputImage[pixelIdx1] / numOfIterations;
	}
}

void STRESSColorToColorCPU3(uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, const uint8_t imageChannels, short int **spraysX, short int **spraysY, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfSprays, const unsigned int numOfIterations) {
	float randomRadius;													// random radius
	float randomTheta;													// random theta
	int samplePixelX;													// (pre-computed / random) sample pixel abscissa
	int samplePixelY;													// (pre-computed / random) sample pixel ordinate
	unsigned int sampleIdx;												// sample index
	uint8_t channelIdx;													// channel index
	unsigned int samplePixelIdx;										// sample pixel index
	unsigned int pixelChannelIdx;										// pixel channel index
	unsigned int numOfValidSamplePoints;								// number of valid sample points in envelope
	std::default_random_engine generator;								// random number generator engine
	std::uniform_real_distribution<float> radiusDistribution(0, radius);	// uniform real distribution for radius in the range (0, radius)
	std::uniform_real_distribution<float> thetaDistribution(0, 2 * M_PI);	// uniform real distribution for theta in the range (0, 2*pi)

	unsigned int targetPixelIdx; // target pixel (p) index
	uint8_t *Emin = (uint8_t*)malloc(imageChannels * sizeof(uint8_t));	// Emin array of size imageChannels
	uint8_t *Emax = (uint8_t*)malloc(imageChannels * sizeof(uint8_t));	// Emax array of size imageChannels

	unsigned int randomSprayIdx;  // random spray index
	short int *randomSprayX;    // abscissas for spray chosen at random
	short int *randomSprayY;    // ordinates for spray chosen at random

	// allocate temporary output image array for storing sum of all iteration results
	unsigned int imageSize = imageWidth * imageHeight * imageChannels;
	float *tempOutputImage = (float*)malloc(imageSize * sizeof(float));

	// initial temporary output image as empty
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		tempOutputImage[pixelIdx] = 0.0f;
	}

	// iteration loop
	for (unsigned int iterationIdx = 0; iterationIdx < numOfIterations; iterationIdx++) {
		targetPixelIdx = 0; // reset target pixel index to 0
		for (unsigned short int targetPixelY = 0; targetPixelY < imageHeight; targetPixelY++) {
			for (unsigned short int targetPixelX = 0; targetPixelX < imageWidth; targetPixelX++) {
				//set Emin and Emax equal to target pixel across all color channels
				for (channelIdx = 0; channelIdx < imageChannels; channelIdx++)
					Emin[channelIdx] = Emax[channelIdx] = inputImage[targetPixelIdx + channelIdx];

				// choose spray at random
				randomSprayIdx = rand() % numOfSprays;
				randomSprayX = spraysX[randomSprayIdx];
				randomSprayY = spraysY[randomSprayIdx];

				// calculate envelope
				sampleIdx = 0;	// reset sample index to 0
				numOfValidSamplePoints = 0;	// reset number of valid sample points to 0
				for (sampleIdx = 0; sampleIdx < numOfSamplePoints; sampleIdx++) {
					samplePixelX = targetPixelX + randomSprayX[sampleIdx];  // get sample pixel abscissa in input image
					samplePixelY = targetPixelY + randomSprayY[sampleIdx];  // get sample pixel ordinate in input image
					if (samplePixelX >= 0 && samplePixelX < imageWidth && samplePixelY >= 0 && samplePixelY < imageHeight) {  // only proceed if sample pixel is within the input image
						samplePixelIdx = (imageWidth * samplePixelY + samplePixelX) * imageChannels; // get sample pixel index in input image
						for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
							pixelChannelIdx = samplePixelIdx + channelIdx;
							if (inputImage[pixelChannelIdx] < Emin[channelIdx]) // if sample pixel value is less than Emin
								Emin[channelIdx] = inputImage[pixelChannelIdx];		// it is the new Emin
							else if (inputImage[pixelChannelIdx] > Emax[channelIdx])	// if sample pixel value is greater than Emax
								Emax[channelIdx] = inputImage[pixelChannelIdx];		// it is the new Emax
						}
						numOfValidSamplePoints++;	// increment number of valid sample points
					}
				}

				// generate sample points to compensate for invalid sample points
				sampleIdx = numOfValidSamplePoints;
				while (sampleIdx < numOfSamplePoints) {
					randomRadius = radiusDistribution(generator);	// get a random distance from the uniform real distribution for distance
					randomTheta = thetaDistribution(generator);		// get a random theta from the uniform real distribution for theta
					samplePixelX = targetPixelX + randomRadius * cos(randomTheta);	// compute random pixel abscissa
					if (samplePixelX >= 0 && samplePixelX < imageWidth) {		// if random pixel abscissa is within image
						samplePixelY = targetPixelY + randomRadius * sin(randomTheta);		// compute random pixel ordinate
						if (samplePixelY >= 0 && samplePixelY < imageHeight) {	// if random pixel ordinate is within image
							samplePixelIdx = imageWidth * samplePixelY + samplePixelX; // get random sample pixel index in image
							for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
								pixelChannelIdx = samplePixelIdx + channelIdx;
								if (inputImage[pixelChannelIdx] < Emin[channelIdx])		// if sample pixel value is less than Emin
									Emin[channelIdx] = inputImage[pixelChannelIdx];		// it is the new Emin
								else if (inputImage[pixelChannelIdx] > Emax[channelIdx])	// if sample pixel value is greater than Emax 
									Emax[channelIdx] = inputImage[pixelChannelIdx];		// it is the new Emax
							}
							sampleIdx++;	// advance random sample pixel index
						}
					}
				}

				// calculate (p - Emin) / (Emax - Emin) for each color channel
				for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
					pixelChannelIdx = targetPixelIdx + channelIdx;
					tempOutputImage[pixelChannelIdx] += (inputImage[pixelChannelIdx] - Emin[channelIdx]) * 255.0 / (Emax[channelIdx] - Emin[channelIdx]);
				}

				targetPixelIdx += imageChannels;
			}
		}
	}

	// divide each accumulated pixel value by the number of iterations to obtain the average pixel value across iterations.
	// place the average value in the output image array.
	for (unsigned int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
		outputImage[pixelIdx] = tempOutputImage[pixelIdx] / numOfIterations;
	}
}


// thanks to http://aresio.blogspot.com/2011/05/cuda-random-numbers-inside-kernels.html
// and to https://hpc.oit.uci.edu/nvidia-doc/sdk-cuda-doc/CUDALibraries/doc/CURAND_Library.pdf
__global__ void setupRandomKernel(hiprandState *state, const unsigned long long seed, const unsigned short int imageWidth, const unsigned short int imageHeight) {
	unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < imageHeight && col < imageWidth) {
		unsigned int idx = imageWidth * row + col; // absolute thread index
		//hiprand_init(seed, idx, 0, &state[idx]);	// initialize random number generator state in global memory
		//hiprand_init((unsigned long long)clock(), 0, 0, &state[idx]);	// initialize random number generator state in global memory
		//hiprand_init(seed, 0, 0, &state[idx]);	// initialize random number generator state in global memory
		//hiprand_init(seed, idx % 2048, 0, &state[idx]);	// initialize random number generator state in global memory
		hiprand_init(seed + idx, 0, 0, &state[idx]);	// initialize random number generator state in global memory
	}
}

__global__ void STRESSColorToGrayscaleKernel2(hiprandState *state, uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, const uint8_t imageChannels, const unsigned int radius, const unsigned int numOfSamplePoints, const unsigned int numOfIterations) {
	unsigned int targetPixelX = blockDim.x * blockIdx.x + threadIdx.x; // get pixel abscissa
	unsigned int targetPixelY = blockDim.y * blockIdx.y + threadIdx.y; // target pixel ordinate

	if (targetPixelX < imageWidth && targetPixelY < imageHeight) {
		unsigned int idx = imageWidth * targetPixelY + targetPixelX;	// thread absolute index (output pixel absolute index)
		hiprandState localState = state[idx];	// load random number generator state from global memory
		
		float randomRadius;		// random radius
		float randomTheta;		// random theta
		const float circle = 2 * M_PI; // 2Pi
		int randomSamplePixelX;	// random sample pixel abscissa
		int randomSamplePixelY;	// random sample pixel ordinate
		unsigned int randomSamplePixelIdx;	// random sample pixel index
		unsigned int randomSampleImagePixelIdx;	// random sample pixel absolute index in image
		unsigned int randomSampleImagePixelChannelIdx;	// random sample pixel channel index

		unsigned int targetPixelIdx = idx * imageChannels;	// target pixel (p) absolute index
		unsigned int targetPixelChannelIdx;		// target pixel channel absolute index
		uint8_t targetPixel[3]; // target pixel array for channels
		uint8_t samplePixel[3];	// sample pixel array for channels
		double outputPixel;		// output pixel values accumulator across iterations
		uint8_t channelIdx;		// channel index
		
		uint8_t Emin[3];	// Emin array of size imageChannels
		uint8_t Emax[3];	// Emax array of size imageChannels
		
		// for calculating (p - Emin).(Emax - Emin) / |Emax - Emin|^2
		uint8_t Edelta;
		unsigned int dotProd, ElenSq;
		
		//outputImage[idx] = 0; // initialize output image as empty
		
		// initialize output pixel values accumulator to 0
		outputPixel = 0.0f;

		// iteration loop
		for (unsigned int iterationIdx = 0; iterationIdx < numOfIterations; iterationIdx++) {
			// load target pixel and set Emin and Emax equal to target pixel value at each channel
			for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
				targetPixelChannelIdx = targetPixelIdx + channelIdx;
				Emin[channelIdx] = Emax[channelIdx] = targetPixel[channelIdx] = inputImage[targetPixelChannelIdx];
			}

			// generate random sample points and calculate envelope
			randomSamplePixelIdx = 0;
			while (randomSamplePixelIdx < numOfSamplePoints) {
				randomRadius = hiprand_uniform(&localState) * radius; // get a random distance from the uniform real distribution
				randomTheta = hiprand_uniform(&localState) * circle; // get a random angle from the uniform real distribution
				randomSamplePixelX = targetPixelX + randomRadius * cosf(randomTheta);	// compute random pixel abscissa
				if (randomSamplePixelX >= 0 && randomSamplePixelX < imageWidth) {	// if random pixel abscissa is within image
					randomSamplePixelY = targetPixelY + randomRadius * sinf(randomTheta);	// compute random pixel ordinate
					if (randomSamplePixelY >= 0 && randomSamplePixelY < imageHeight) {	// if random pixel ordinate is within image
						randomSampleImagePixelIdx = (imageWidth * randomSamplePixelY + randomSamplePixelX) * imageChannels;	// get random sample pixel index in image
						for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
							randomSampleImagePixelChannelIdx = randomSampleImagePixelIdx + channelIdx;	// get random sample pixel channel index
							samplePixel[channelIdx] = inputImage[randomSampleImagePixelChannelIdx];
							if (samplePixel[channelIdx] < Emin[channelIdx])			// if random sample pixel channel value is less than Emin at that channel
								Emin[channelIdx] = samplePixel[channelIdx];			// it is the new Emin
							else if (samplePixel[channelIdx] > Emax[channelIdx])	// if random sample pixel channel value is greater than Emax at that channel
								Emax[channelIdx] = samplePixel[channelIdx];			// it is the new Emax
						}
						randomSamplePixelIdx++;	// advance random sample pixel index
					}
				}
			}

			dotProd = 0;
			ElenSq = 0;
			for (channelIdx = 0; channelIdx < imageChannels; channelIdx++) {
				Edelta = Emax[channelIdx] - Emin[channelIdx];
				dotProd += Edelta * (targetPixel[channelIdx] - Emin[channelIdx]);
				ElenSq += Edelta * Edelta;
			}

			// calculate g = (p - Emin).(Emax - Emin) / |Emax - Emin|^2
			outputPixel += dotProd * 255.0 / ElenSq;
		}
		outputImage[idx] = outputPixel / numOfIterations;
		state[idx] = localState;	// store updated random number generator state back into global memory
	}
}




int main(int argc, char *argv[])
{
	if (argc != 7) {
		fprintf(stderr, "Invalid number of arguments.");
		return 1;
	}

	srand(time(NULL));
	const unsigned short int radius = atoi(argv[3]);
	const unsigned int numOfSamplePoints = atoi(argv[4]);
	const unsigned int numOfIterations = atoi(argv[5]);
	const unsigned int numOfSprays = atoi(argv[6]);
	short int **spraysX;
	short int **spraysY;
	clock_t computeRandomSpraysCPUClock = clock();
	computeRandomSpraysCPU(&spraysX, &spraysY, radius, numOfSamplePoints, numOfSprays);
	double computeRandomSpraysCPUDuration = (clock() - computeRandomSpraysCPUClock) / (double)CLOCKS_PER_SEC;

	printf("Time to compute random sprays (CPU): %fs\n", computeRandomSpraysCPUDuration);

	/*printf("Writing random sprays (%i) to disk ...\n", numOfSprays);
	char sprayImageName[20];
	for (unsigned int sprayIdx = 0; sprayIdx < numOfSprays; sprayIdx++) {
		cv::Mat sprayImage = generateRandomSprayImage(spraysX[sprayIdx], spraysY[sprayIdx], radius, numOfSamplePoints);
		sprintf(sprayImageName, "spray%i.png", sprayIdx);
		cv::imwrite(sprayImageName, sprayImage);
	}*/

	char imageName[50];

	/*char *grayscaleImageName = argv[1];
	cv::Mat grayscaleInputImage = cv::imread(grayscaleImageName, CV_LOAD_IMAGE_GRAYSCALE);
	if (grayscaleInputImage.empty()) {
		fprintf(stderr, "Cannot read grayscale image file %s.", grayscaleImageName);
		return 1;
	}
	uint8_t *grayscaleOutputImageData = (uint8_t*)malloc(grayscaleInputImage.cols * grayscaleInputImage.rows * sizeof(uint8_t));

	printf("Running STRESSGrayscaleToGrayscaleCPU1 (R=%i, M=%i, N=%i, S=%i) ...\n", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	clock_t STRESSG2GCPU1Clock = clock();
	STRESSGrayscaleToGrayscaleCPU1(grayscaleOutputImageData, grayscaleInputImage.data, grayscaleInputImage.cols, grayscaleInputImage.rows, spraysX, spraysY, numOfSamplePoints, numOfSprays, numOfIterations);
	double STRESSG2GCPU1Duration = (clock() - STRESSG2GCPU1Clock) / (double) CLOCKS_PER_SEC;
	printf("Finished STRESSGrayscaleToGrayscaleCPU1 in %fs, dumping to disk ...\n", STRESSG2GCPU1Duration);
	cv::Mat G2GOutputImageCPU1(grayscaleInputImage.rows, grayscaleInputImage.cols, CV_8UC1, grayscaleOutputImageData);
	sprintf(imageName, "outG2GCPU1_R%i_M%i_N%i_S%i.png", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	cv::imwrite(imageName, G2GOutputImageCPU1);

	printf("Running STRESSGrayscaleToGrayscaleCPU2 (R=%i, M=%i, N=%i) ...\n", radius, numOfSamplePoints, numOfIterations);
	clock_t STRESSG2GCPU2Clock = clock();
	STRESSGrayscaleToGrayscaleCPU2(grayscaleOutputImageData, grayscaleInputImage.data, grayscaleInputImage.cols, grayscaleInputImage.rows, radius, numOfSamplePoints, numOfIterations);
	double STRESSG2GCPU2Duration = (clock() - STRESSG2GCPU2Clock) / (double)CLOCKS_PER_SEC;
	printf("Finished STRESSGrayscaleToGrayscaleCPU2 in %fs, dumping to disk ...\n", STRESSG2GCPU2Duration);
	cv::Mat G2GOutputImageCPU2(grayscaleInputImage.rows, grayscaleInputImage.cols, CV_8UC1, grayscaleOutputImageData);
	sprintf(imageName, "outG2GCPU2_R%i_M%i_N%i_S%i.png", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	cv::imwrite(imageName, G2GOutputImageCPU2);

	printf("Running STRESSGrayscaleToGrayscaleCPU3 (R=%i, M=%i, N=%i, S=%i) ...\n", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	clock_t STRESSG2GCPU3Clock = clock();
	STRESSGrayscaleToGrayscaleCPU3(grayscaleOutputImageData, grayscaleInputImage.data, grayscaleInputImage.cols, grayscaleInputImage.rows, spraysX, spraysY, radius, numOfSamplePoints, numOfSprays, numOfIterations);
	double STRESSG2GCPU3Duration = (clock() - STRESSG2GCPU3Clock) / (double)CLOCKS_PER_SEC;
	printf("Finished STRESSGrayscaleToGrayscaleCPU3 in %fs, dumping to disk ...\n", STRESSG2GCPU3Duration);
	cv::Mat G2GOutputImageCPU3(grayscaleInputImage.rows, grayscaleInputImage.cols, CV_8UC1, grayscaleOutputImageData);
	sprintf(imageName, "outG2GCPU3_R%i_M%i_N%i_S%i.png", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	cv::imwrite(imageName, G2GOutputImageCPU3);*/
	
	char *colorimageName = argv[2];
	cv::Mat colorInputImage = cv::imread(colorimageName, CV_LOAD_IMAGE_COLOR);
	uint8_t *C2GOutputImageData = (uint8_t*)malloc(colorInputImage.cols * colorInputImage.rows * sizeof(uint8_t));

	printf("Running STRESSColorToGrayscaleCPU3 (R=%i, M=%i, N=%i, S=%i) ...\n", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	clock_t STRESSC2GCPU3Clock = clock();
	STRESSColorToGrayscaleCPU3(C2GOutputImageData, colorInputImage.data, colorInputImage.cols, colorInputImage.rows, colorInputImage.channels(), spraysX, spraysY, radius, numOfSamplePoints, numOfSprays, numOfIterations);
	double STRESSC2GCPU3Duration = (clock() - STRESSC2GCPU3Clock) / (double)CLOCKS_PER_SEC;
	printf("Finished STRESSColorToGrayscaleCPU3 in %fs, dumping to disk ...\n", STRESSC2GCPU3Duration);
	cv::Mat C2GOutputImageCPU3(colorInputImage.rows, colorInputImage.cols, CV_8UC1, C2GOutputImageData);
	sprintf(imageName, "outC2GCPU3_R%i_M%i_N%i_S%i.png", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	cv::imwrite(imageName, C2GOutputImageCPU3);

	/*unsigned int colorImageSize = colorInputImage.cols * colorInputImage.rows * colorInputImage.channels();
	uint8_t *colorOutputImageData = (uint8_t*)malloc(colorImageSize * sizeof(uint8_t));

	printf("Running STRESSColorToColorCPU3 (R=%i, M=%i, N=%i, S=%i) ...\n", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	clock_t STRESSC2CCPU3Clock = clock();
	STRESSColorToColorCPU3(colorOutputImageData, colorInputImage.data, colorInputImage.cols, colorInputImage.rows, colorInputImage.channels(), spraysX, spraysY, radius, numOfSamplePoints, numOfSprays, numOfIterations);
	double STRESSC2CCPU3Duration = (clock() - STRESSC2CCPU3Clock) / (double)CLOCKS_PER_SEC;
	printf("Finished STRESSColorToColorCPU3 in %fs, dumping to disk ...\n", STRESSC2CCPU3Duration);
	cv::Mat C2COutputImageCPU3(colorInputImage.rows, colorInputImage.cols, CV_8UC3, colorOutputImageData);
	sprintf(imageName, "outC2CCPU3_R%i_M%i_N%i_S%i.png", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	cv::imwrite(imageName, C2COutputImageCPU3);*/

	printf("Running STRESSColorToGrayscaleKernel2 (R=%i, M=%i, N=%i, S=%i) ...\n", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	unsigned long seed = time(NULL);
	hipError_t cudaStatus = testWithCuda(C2GOutputImageData, colorInputImage.data, colorInputImage.cols, colorInputImage.rows, colorInputImage.channels(), spraysX, spraysY, radius, numOfSamplePoints, numOfSprays, numOfIterations, seed);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "testWithCuda failed!");
        return 1;
    }
	printf("Finished STRESSColorToGrayscaleKernel2, dumping to disk ...\n");
	cv::Mat C2GOutputImageGPU2(colorInputImage.rows, colorInputImage.cols, CV_8UC1, C2GOutputImageData);
	sprintf(imageName, "outC2GGPU2_R%i_M%i_N%i_S%i.png", radius, numOfSamplePoints, numOfIterations, numOfSprays);
	cv::imwrite(imageName, C2GOutputImageGPU2);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	system("PAUSE");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t testWithCuda(uint8_t *outputImage, uint8_t *inputImage, const unsigned short int imageWidth, const unsigned short int imageHeight, const uint8_t imageChannels, short int **spraysX, short int **spraysY, const unsigned short int radius, const unsigned int numOfSamplePoints, const unsigned int numOfSprays, const unsigned int numOfIterations, const unsigned long long seed)
{
	GpuTimer cudaMallocInputTimer;
	GpuTimer cudaMallocOutputTimer;
	GpuTimer cudaMallocCurandStatesTimer;
	GpuTimer cudaMemcpyInputTimer;
	GpuTimer cudaSetupRandomKernelTimer;
	GpuTimer cudaSTRESSColorToGrayscaleKernelTimer;
	GpuTimer cudaMemcpyOutputTimer;
	unsigned int outputImageSize = imageWidth * imageHeight;
	unsigned int inputImageSize = outputImageSize * imageChannels;
	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for two vectors (one input, one output).
	uint8_t *d_InputImage;
	cudaMallocInputTimer.Start();
    cudaStatus = hipMalloc((void**)&d_InputImage, inputImageSize * sizeof(uint8_t));
	cudaMallocInputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc (input image) failed!");
        goto Error;
    }
	printf("Time to allocate input:\t\t\t\t\t%f ms\n", cudaMallocInputTimer.Elapsed());

	
	uint8_t *d_OutputImage;
	cudaMallocOutputTimer.Start();
    cudaStatus = hipMalloc((void**)&d_OutputImage, outputImageSize * sizeof(uint8_t));
	cudaMallocOutputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc (output image) failed!");
        goto Error;
    }
	printf("Time to allocate output:\t\t\t\t%f ms\n", cudaMallocOutputTimer.Elapsed());

	// Declare block and grid dimensions
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	unsigned int gridDimX = (imageWidth - 1) / BLOCK_WIDTH + 1;
	unsigned int gridDimY = (imageHeight - 1) / BLOCK_WIDTH + 1;
	dim3 dimGrid(gridDimX, gridDimY, 1);

	// Allocate random number generator states
	//unsigned int numOfThreads = gridDimX * gridDimY * BLOCK_WIDTH * BLOCK_WIDTH;
	hiprandState *d_CURANDStates;
	cudaMallocCurandStatesTimer.Start();
	cudaStatus = hipMalloc((void**)&d_CURANDStates, outputImageSize * sizeof(hiprandState));
	cudaMallocCurandStatesTimer.Stop();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc (CURAND states) failed!");
		goto Error;
	}
	printf("Time to allocate CURAND states:\t\t\t\t%f ms\n", cudaMallocCurandStatesTimer.Elapsed());

	// Launch the setup random number generator kernel on the GPU with one thread for each element.
	cudaSetupRandomKernelTimer.Start();
	setupRandomKernel <<<dimGrid, dimBlock>>>(d_CURANDStates, seed, imageWidth, imageHeight);
	cudaSetupRandomKernelTimer.Stop();

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "setupRandomKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching setupRandomKernel!\n", cudaStatus);
		goto Error;
	}
	printf("Time to execute setupRandomKernel kernel:\t\t%f ms\n", cudaSetupRandomKernelTimer.Elapsed());

	// Copy input vectors from host memory to GPU buffers.
	cudaMemcpyInputTimer.Start();
	cudaStatus = hipMemcpy(d_InputImage, inputImage, inputImageSize * sizeof(uint8_t), hipMemcpyHostToDevice);
	cudaMemcpyInputTimer.Stop();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy (host -> device) failed!");
		goto Error;
	}
	printf("Time to copy input from host to device:\t\t\t%f ms\n", cudaMemcpyInputTimer.Elapsed());

	// Launch the STRESS color to grayscale kernel on the GPU with one thread for each element.
	cudaSTRESSColorToGrayscaleKernelTimer.Start();
    STRESSColorToGrayscaleKernel2<<<dimGrid, dimBlock>>>(d_CURANDStates, d_OutputImage, d_InputImage, imageWidth, imageHeight, imageChannels, radius, numOfSamplePoints, numOfIterations);
	cudaSTRESSColorToGrayscaleKernelTimer.Stop();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "STRESSColorToGrayscaleKernel2 launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching STRESSColorToGrayscaleKernel2!\n", cudaStatus);
        goto Error;
    }
	printf("Time to execute STRESSColorToGrayscaleKernel2 kernel:\t%f ms\n", cudaSTRESSColorToGrayscaleKernelTimer.Elapsed());

    // Copy output vector from GPU buffer to host memory.
	cudaMemcpyOutputTimer.Start();
    cudaStatus = hipMemcpy(outputImage, d_OutputImage, outputImageSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
	cudaMemcpyOutputTimer.Stop();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy (device -> host) failed!");
        goto Error;
    }

	{
		printf("Time to copy output from device to host:\t\t%f ms\n", cudaMemcpyOutputTimer.Elapsed());
	}


Error:
	hipFree(d_CURANDStates);
	hipFree(d_InputImage);
    hipFree(d_OutputImage);
    
	return cudaStatus;
}
